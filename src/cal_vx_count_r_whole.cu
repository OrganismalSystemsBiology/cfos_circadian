#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>    
#include "common.h"  //-I/opt/nvidia/hpc_sdk/Linux_x86_64/23.3/examples/OpenACC/SDK/include/    ~.h

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <limits.h>
#include <vector>
#include <string>
#include <array>
#include <math.h>
#include <tuple>
#include <time.h>
#include <cmath>
#include <algorithm>
#include <numeric>
#include <stdlib.h>
#include "numpy.hpp"

#include <filesystem>  
#include <random>
#include <omp.h>  //OpenMP

//for voxel based phase in whole brain
//nvc++ -mp -ta=tesla cal_vx_count_r_whole.cu -I/opt/nvidia/hpc_sdk/Linux_x86_64/23.3/examples/OpenACC/SDK/include/ -I/tmp/nvhpc_2023_233_Linux_x86_64_cuda_multi/install_components/Linux_x86_64/23.3/math_libs/12.0/targets/x86_64-linux/include -L/tmp/nvhpc_2023_233_Linux_x86_64_cuda_multi/install_components/Linux_x86_64/23.3/math_libs/12.0/targets/x86_64-linux/lib -lcudart -std=c++20 -o vx_c_w
//./vx_c_w 2 2 2 144 197 175  SCH 50 3 1 0.5 /home/gpu_data/data8/cfos_app ANTsR50 2 2 2
//./vx_cr 153 163 82 114 165 157 SCH 50 3 1 0.5 /home/gpu_data/data8/cfos_app ANTsR50 2 8 8 8

//CUDA error check
#define CHECK(call)                   \
{                                      \
  const hipError_t error = call;      \
  if(error != hipSuccess)             \
  {                                    \
      printf("Error: %s:%d, ", __FILE__, __LINE__);               \
      printf("code:%d, reason: %s\n", error, hipGetErrorString(error));    \
      exit(1);        \
  }                \
}             \



double cpuSecond(){
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}


__global__ void vx_gpu(int* vxx, int* vxy,  int* vxz, int vx_num, int *x, int *y,  int *z, int *vx_count, int cell_count,  int vx, int vb_r){

    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    // unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    // unsigned int iz = threadIdx.z + blockIdx.z * blockDim.z;
    // unsigned int idx =  ix *y_b_num * z_b_num +  iy * z_b_num + iz;

    if (ix < vx_num) {

                int count=0;
                // printf("count %d, N %d", ix, iy);
                for (int it=0; it<cell_count; it++){
                    // if ((ix<1)&(iy<1)&(iy<1)&(it<3)){
                    // printf("it %d, x %d, y %d, z %d \n", it, x[it], y[it], z[it]);
                    // printf("xrange low:%d\n", xmin*vx+ix*mo*vx-int(vb_r*vx/2));
                    // printf("xrange high:%d\n", xmin*vx+ix*mo*vx+int(vb_r*vx/2));
                    // }
                    if((x[it]>=vxx[ix]*vx-int(vb_r*vx/2))&(x[it]<vxx[ix]*vx+int(vb_r*vx/2))&(y[it]>=vxy[ix]*vx-int(vb_r*vx/2))&(y[it]<vxy[ix]*vx+int(vb_r*vx/2))&(z[it]>=vxz[ix]*vx-int(vb_r*vx/2))&(z[it]<vxz[ix]*vx+int(vb_r*vx/2))){
                    count+=1;
                      }//if(x~)
                    } //for  cell_count
                vx_count[ix] = count;
                // if ((ix<3)&(iy<3)&(iy<3)){
                // printf("count %d\n", count);
                // printf("idx %d\n", idx);
                // }
            }
        }


std::string getLeftPaddingString(std::string const &str, int n, char paddedChar = ' ')
{
    std::ostringstream ss;
    ss << std::right << std::setfill(paddedChar) << std::setw(n) << str;
    return ss.str();
}

bool fileExists(const std::string& path) {
    return std::filesystem::exists(path);
}



int main(int argc, char * argv[]){
int g_c=1;
//importing bin file

// int cell_count = atoi(argv[g_c]); 
// g_c +=1;

// std::string region = argv[g_c];
// g_c +=1;
int vx = atoi(argv[g_c]); 
g_c +=1;
int vb_r = atoi(argv[g_c]); 
g_c +=1;
int mo = atoi(argv[g_c]); 
g_c +=1;

std::string vx_cords_f = argv[g_c];
g_c +=1;

// std::cout << r_str << std::endl;
// printf("r %s\n", r_str.c_str());

std::string savedir = argv[g_c];
g_c +=1;

std::string root_fol = argv[g_c];
g_c +=1;

std::string ants_dir_name = argv[g_c];
g_c +=1;

std::string combine_points_f = argv[g_c];//"cell_table_combine.npy";
g_c+=1;

int cpu_num = atoi(argv[g_c]);
g_c +=1;

printf("vb_r,  %d\n", vb_r);
printf("ants file %s\n", ants_dir_name.c_str());


int blockdim_x =atoi(argv[g_c]); //cp
g_c +=1;


printf("blockdim_x  %d\n", blockdim_x);

//read cell coordinates file
int CT_num = 48;
std::vector<int> CT_li;
for (int i = 0; i < CT_num ; i += 4) {
    CT_li.push_back(i);
}

std::vector<int> sample_ids;
for (int i = 1; i < 7 ; i += 1) {
    sample_ids.push_back(i);
}

// std::string sample = "CT0_01";
std::vector<std::string> exps{"1st", "2nd"} ;

//start timer
double iStart = cpuSecond();
//GPU device setup
int dev = 0;
hipDeviceProp_t deviceProp;
CHECK(hipGetDeviceProperties(&deviceProp, dev));
printf("Using Device %d: %s\n", dev, deviceProp.name);
CHECK(hipSetDevice(dev));

for (int l = 0; l < exps.size() ; l += 1) {
   
    std::string exp = exps[l];
    printf(" %s\n", exp.c_str());

    #pragma omp parallel for num_threads(cpu_num)
    for (int m = 0; m < CT_li.size() ; m += 1) {
        
        for (int n = 0; n < sample_ids.size() ; n += 1) {
            std::string pd = getLeftPaddingString(std::to_string(sample_ids[n]), 2, '0')  ;

            std::string sample = "CT"+std::to_string(CT_li[m])+ "_" + pd;
            printf(" %s\n", sample.c_str());

            std::string root_vx_f = savedir + "/" +exp + "/"+root_fol+"/"+std::to_string(vx)+"um/whole/vb"+ std::to_string(vb_r)+"_mo"+std::to_string(mo)+"/";
            std::filesystem::create_directories(root_vx_f); 

            std::string vx_f= root_vx_f + sample + "_vb_CT_"+exp+".bin";

            if (fileExists(vx_f)) {
                std::cout << vx_f << " exists" << std::endl;
            }else{

            //voxel coordinates from atlas
            std::vector<int> s0;
            std::vector<int> vx_cords;  
            std::string vx_npy = vx_cords_f;  // vx coordinates of brains

            std::cout << vx_npy << std::endl;

            aoba::LoadArrayFromNumpy(vx_npy, s0, vx_cords);
            std::cout << "size:" << s0[0] << " " << s0[1] << std::endl;

            printf("i: %d ,  vx_cords[0]:%d\n", 0, vx_cords[0]);

            int vx_num = s0[1];
            int col_num_vx = s0[0];
            printf("total voxel  %d\n", vx_num);
            std::vector<int> vxx(sizeof(int)*vx_num);
            std::vector<int> vxy(sizeof(int)*vx_num);
            std::vector<int> vxz(sizeof(int)*vx_num);


                for (int i = 0; i < vx_num ; i += 1) {
                    for (int j = 0; j < col_num_vx ; j += 1) {
                    if (j==0){
                    vxx[i] = vx_cords[j*vx_num+i];
                    }else if(j==1){
                    vxy[i] = vx_cords[j*vx_num+i];
                    }else if(j==2){
                    vxz[i] = vx_cords[j*vx_num+i];
                    }
                    if (i<3){
                    printf("i: %d ,  vxx[i]:%d\n", i, vxx[i]);
                    printf("i: %d ,  vxy[i]:%d\n", i, vxy[i]);
                    printf("i: %d ,  xvz[i]:%d\n", i, vxz[i]);
                    }
                }
                }  




            std::vector<int> s;
            std::vector<double> cell_cords;  
            std::string cell_npy = savedir +"/"+ exp+"/" +  sample +"/SYTOX-G/"+ants_dir_name +"/"+combine_points_f;
            aoba::LoadArrayFromNumpy(cell_npy, s, cell_cords);
            std::cout << "size:" << s[0] << " " << s[1] << std::endl;

            printf("i: %d ,  cell_cords[0]:%f\n", 0, cell_cords[0]);

            int cell_count = s[1];
            int col_num = s[0];

            printf("col_num,  %d\n", col_num);
            printf("cell_count,  %d\n", cell_count);

            // int *x=(int*)malloc(sizeof(int)*cell_count);
            // int *y=(int*)malloc(sizeof(int)*cell_count);
            // int *z=(int*)malloc(sizeof(int)*cell_count);

            std::vector<int> x(sizeof(int)*cell_count);
            std::vector<int> y(sizeof(int)*cell_count);
            std::vector<int> z(sizeof(int)*cell_count);


            for (int i = 0; i < cell_count ; i += 1) {
                for (int j = 0; j < col_num ; j += 1) {
                if (j==0){
                x[i] = static_cast<int>(cell_cords[j*cell_count+i]);
                }else if(j==1){
                y[i] = static_cast<int>(cell_cords[j*cell_count+i]);
                }else if(j==2){
                z[i] = static_cast<int>(cell_cords[j*cell_count+i]);
                }

                // if (i<3){
                // printf("i: %d ,  x[i]:%d\n", i, x[i]);
                // printf("i: %d ,  y[i]:%d\n", i, y[i]);
                // printf("i: %d ,  z[i]:%d\n", i, z[i]);
                // }
            }
            }

            if ((l==0)&(m==0)&(n==0)){
                for (int i = 0; i < 2 ; i += 1) {
                    printf("i: %d ,  x[i]:%d\n", i, x[i]);
                    printf("i: %d ,  y[i]:%d\n", i, y[i]);
                    printf("i: %d ,  z[i]:%d\n", i, z[i]);
                }
            }



            int *vx_count=(int*)malloc(sizeof(int)*vx_num);
            // double *vx_count_ratio=(double*)malloc(sizeof(double)*vx_num);

            // printf("a\n");
            // 


            // printf("aa\n");
            //device memory
            int *d_vxx,  *d_vxy, *d_vxz;
            CHECK(hipMalloc((void **)&d_vxx, sizeof(int)*vx_num));
            CHECK(hipMalloc((void **)&d_vxy, sizeof(int)*vx_num));
            CHECK(hipMalloc((void **)&d_vxz, sizeof(int)*vx_num));

            int *d_x,  *d_y, *d_z, *d_vx_count;
            CHECK(hipMalloc((void **)&d_x, sizeof(int)*cell_count));
            CHECK(hipMalloc((void **)&d_y, sizeof(int)*cell_count));
            CHECK(hipMalloc((void **)&d_z, sizeof(int)*cell_count));
            CHECK(hipMalloc((void **)&d_vx_count, sizeof(int)*vx_num));


            // printf("aaa\n");

            //memcopy host -> device
            CHECK(hipMemcpy(d_vxx, vxx.data(), sizeof(int)*vx_num, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_vxy, vxy.data(), sizeof(int)*vx_num, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_vxz, vxz.data(), sizeof(int)*vx_num, hipMemcpyHostToDevice));    
            

            CHECK(hipMemcpy(d_x, x.data(), sizeof(int)*cell_count, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_y, y.data(), sizeof(int)*cell_count, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_z, z.data(), sizeof(int)*cell_count, hipMemcpyHostToDevice));    
            CHECK(hipMemcpy(d_vx_count, vx_count, sizeof(int)*vx_num, hipMemcpyHostToDevice));   

               

                int dimx =vx_num/blockdim_x;// (NE+NI)/blockdim_x;
                // int dimy = y_b_num/blockdim_y;
                // int dimz = z_b_num/blockdim_z;
                dim3 block(blockdim_x);
                dim3 grid(dimx);

                

                vx_gpu<<<grid, block>>>(d_vxx, d_vxy,  d_vxz, vx_num, d_x, d_y,  d_z, d_vx_count,  cell_count,  vx, vb_r);
                CHECK(hipDeviceSynchronize());
                CHECK(hipGetLastError());
                printf("end vx_count\n");
            
            

                        // copy kernel result back to host side
                
                CHECK(hipMemcpy(vx_count, d_vx_count, sizeof(int)*vx_num,  hipMemcpyDeviceToHost));


                 for (int i=0; i<vx_num; i++){
                    // vx_count_ratio[i] = static_cast<int>(vx_count[i]/cell_count);
                    if ((i<3)&(l==0)&(m==0)&(n==0)){
                    printf("i: %d ,  vx_count[i]:%d\n", i, vx_count[i]);
                    }
                 }
                
            //savefile
                
            //savefiles
            
            
            std::ofstream ofs;
                ofs.open(vx_f, std::ios::out|std::ios::binary|std::ios::trunc);
                if (!ofs) {
                std::cout << "Can't open a file"<<vx_f<<std::endl;
                }
                    
                for (int i=0; i<vx_num; i++){
                    ofs.write(( char * ) &vx_count[i],sizeof(int) );
                    // printf("count %d, cv %f\n", i, cvs[i]);
                    }//for
                ofs.close();

            CHECK(hipFree(d_x));
            CHECK(hipFree(d_y));
            CHECK(hipFree(d_z));
            CHECK(hipFree(d_vx_count));

            // free(x);
            // free(y);
            // free(z);
            free(vx_count);
            // free(vx_count_ratio);
// free(spc);

        }
    }
   }
}
   //end timer
    double iElaps = cpuSecond() - iStart;
    printf("elapsed %f sec\n", iElaps);
}






// 

