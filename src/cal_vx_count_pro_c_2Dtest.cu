#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>    
#include "common.h"  //-I/opt/nvidia/hpc_sdk/Linux_x86_64/23.3/examples/OpenACC/SDK/include/    ~.h

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <limits.h>
#include <vector>
#include <string>
#include <array>
#include <math.h>
#include <tuple>
#include <time.h>
#include <cmath>
#include <algorithm>
#include <numeric>
#include <stdlib.h>
#include "numpy.hpp"

#include <filesystem>  
#include <random>
#include <omp.h>  //OpenMP


//nvc++ -mp -ta=tesla cal_vx_count_pro_c_2Dtest.cu -I/opt/nvidia/hpc_sdk/Linux_x86_64/23.3/examples/OpenACC/SDK/include/ -I/tmp/nvhpc_2023_233_Linux_x86_64_cuda_multi/install_components/Linux_x86_64/23.3/math_libs/12.0/targets/x86_64-linux/include -L/tmp/nvhpc_2023_233_Linux_x86_64_cuda_multi/install_components/Linux_x86_64/23.3/math_libs/12.0/targets/x86_64-linux/lib -lcudart -std=c++20 -o vx_pro_c_2D
//./vx_pro_c 2 2 2 144 197 175  SCH 50 3 1 0.5 /home/gpu_data/data8/cfos_app ANTsR50 2 2 2
//./vx_pro_c 153 163 82 114 165 157 SCH 50 3 1 0.5 /home/gpu_data/data8/cfos_app ANTsR50 cell_intensity_norm.npy 2 8 8 8

//CUDA error check
#define CHECK(call)                   \
{                                      \
  const hipError_t error = call;      \
  if(error != hipSuccess)             \
  {                                    \
      printf("Error: %s:%d, ", __FILE__, __LINE__);               \
      printf("code:%d, reason: %s\n", error, hipGetErrorString(error));    \
      exit(1);        \
  }                \
}             \



double cpuSecond(){
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}


__global__ void vx_gpu_hor(int *x, int *y,  int *z,  int *vx_count, int cell_count, int x_b_num, int y_b_num, int z_b_num, int xmin, int ymin, int zmin, int xmax, int ymax, int zmax,  int vx, int vb_r, int mox, int moy, int moz){

    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int iz = threadIdx.z + blockIdx.z * blockDim.z;
    unsigned int idx =  ix *y_b_num * z_b_num +  iy * z_b_num + iz;

    if (ix < x_b_num) {
     if (iy < y_b_num) {
        if (iz < z_b_num) {
                int c=0;
                // printf("count %d, N %d", ix, iy);
                for (int it=0; it<cell_count; it++){
                    // if ((ix<1)&(iy<1)&(iy<1)&(it<3)){
                    // printf("it %d, x %d, y %d, z %d \n", it, x[it], y[it], z[it]);
                    // printf("xrange low:%d\n", xmin*vx+ix*mo*vx-int(vb_r*vx/2));
                    // printf("xrange high:%d\n", xmin*vx+ix*mo*vx+int(vb_r*vx/2));
                    // }
                    if((x[it]>=xmin*vx+ix*mox*vx-int(vb_r*vx/2))&(x[it]<xmin*vx+ix*mox*vx+int(vb_r*vx/2))&(y[it]>=ymin*vx+iy*moy*vx-int(vb_r*vx/2))&(y[it]<ymin*vx+iy*moy*vx+int(vb_r*vx/2))&(z[it]>=zmin*vx-int(vb_r*vx/2))&(z[it]<zmax*vx+int(vb_r*vx/2))){
                    c+=1;
                      }//if(x~)
                    } //for  cell_count
                vx_count[idx] = c;
                // if ((ix<3)&(iy<3)&(iy<3)){
                // printf("count %d\n", count);
                // printf("idx %d\n", idx);
                // }
            }
        }
    }
}

__global__ void vx_gpu_cor(int *x, int *y,  int *z, int *vx_count, int cell_count, int x_b_num, int y_b_num, int z_b_num, int xmin, int ymin, int zmin, int xmax, int ymax, int zmax,  int vx, int vb_r, int mox, int moy, int moz){

    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int iz = threadIdx.z + blockIdx.z * blockDim.z;
    unsigned int idx =  ix *y_b_num * z_b_num +  iy * z_b_num + iz;

    if (ix < x_b_num) {
     if (iy < y_b_num) {
        if (iz < z_b_num) {
                int sum=0;
                // printf("count %d, N %d", ix, iy);
                for (int it=0; it<cell_count; it++){
                    // if ((ix<1)&(iy<1)&(iy<1)&(it<3)){
                    // printf("it %d, x %d, y %d, z %d \n", it, x[it], y[it], z[it]);
                    // printf("xrange low:%d\n", xmin*vx+ix*mo*vx-int(vb_r*vx/2));
                    // printf("xrange high:%d\n", xmin*vx+ix*mo*vx+int(vb_r*vx/2));
                    // }
                    if((x[it]>=xmin*vx+ix*mox*vx-int(vb_r*vx/2))&(x[it]<xmin*vx+ix*mox*vx+int(vb_r*vx/2))&(y[it]>=ymin*vx)&(y[it]<ymax*vx)&(z[it]>=zmin*vx+iz*moz*vx-int(vb_r*vx/2))&(z[it]<zmin*vx+iz*moz*vx+int(vb_r*vx/2))){
                    
                    sum+=1;
                      }//if(x~)
                    } //for  cell_count
                vx_count[idx] = sum;
                // if ((ix<3)&(iy<3)&(iy<3)){
                // printf("count %d\n", count);
                // printf("idx %d\n", idx);
                // }
            }
        }
    }
}

__global__ void vx_gpu_sag(int *x, int *y,  int *z, int *vx_count, int cell_count, int x_b_num, int y_b_num, int z_b_num, int xmin, int ymin, int zmin, int xmax, int ymax, int zmax,  int vx, int vb_r, int mox, int moy, int moz){

    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int iz = threadIdx.z + blockIdx.z * blockDim.z;
    unsigned int idx =  ix *y_b_num * z_b_num +  iy * z_b_num + iz;

    if (ix < x_b_num) {
     if (iy < y_b_num) {
        if (iz < z_b_num) {
                int sum=0;
                // printf("count %d, N %d", ix, iy);
                for (int it=0; it<cell_count; it++){
                    // if ((ix<1)&(iy<1)&(iy<1)&(it<3)){
                    // printf("it %d, x %d, y %d, z %d \n", it, x[it], y[it], z[it]);
                    // printf("xrange low:%d\n", xmin*vx+ix*mo*vx-int(vb_r*vx/2));
                    // printf("xrange high:%d\n", xmin*vx+ix*mo*vx+int(vb_r*vx/2));
                    // }
                    if((x[it]>=xmin*vx)&(x[it]<xmax*vx)&(y[it]>=ymin*vx+iy*moy*vx-int(vb_r*vx/2))&(y[it]<ymin*vx+iy*moy*vx+int(vb_r*vx/2))&(z[it]>=zmin*vx+iz*moz*vx-int(vb_r*vx/2))&(z[it]<zmin*vx+iz*moz*vx+int(vb_r*vx/2))){
                    
                    sum+=1;
                      }//if(x~)
                    } //for  cell_count
                vx_count[idx] = sum;
                // if ((ix<3)&(iy<3)&(iy<3)){
                // printf("count %d\n", count);
                // printf("idx %d\n", idx);
                // }
            }
        }
    }
}

std::string getLeftPaddingString(std::string const &str, int n, char paddedChar = ' ')
{
    std::ostringstream ss;
    ss << std::right << std::setfill(paddedChar) << std::setw(n) << str;
    return ss.str();
}

bool fileExists(const std::string& path) {
    return std::filesystem::exists(path);
}


int main(int argc, char * argv[]){
int g_c=1;
//importing bin file

// int cell_count = atoi(argv[g_c]); 
// g_c +=1;

int x_b_num = atoi(argv[g_c]); 
g_c +=1;
int y_b_num = atoi(argv[g_c]); 
g_c +=1;
int z_b_num = atoi(argv[g_c]); 
g_c +=1;

int xmin  = atoi(argv[g_c]); 
g_c +=1;
int ymin = atoi(argv[g_c]); 
g_c +=1;
int zmin  = atoi(argv[g_c]); 
g_c +=1;

int xmax  = atoi(argv[g_c]); 
g_c +=1;
int ymax = atoi(argv[g_c]); 
g_c +=1;
int zmax  = atoi(argv[g_c]); 
g_c +=1;

std::string region = argv[g_c];
g_c +=1;
int vx = atoi(argv[g_c]); 
g_c +=1;
int vb_r = atoi(argv[g_c]); 
g_c +=1;
int mox = atoi(argv[g_c]); 
g_c +=1;
int moy = atoi(argv[g_c]); 
g_c +=1;
int moz = atoi(argv[g_c]); 
g_c +=1;
float r = atof(argv[g_c]); 
std::string r_str = argv[g_c];
g_c +=1;

std::string angle = argv[g_c];
g_c +=1;

// std::cout << r_str << std::endl;
printf("r %s\n", r_str.c_str());

std::string savedir = argv[g_c];
g_c +=1;

std::string root_fol = argv[g_c];
g_c +=1;

std::string ants_dir_name = argv[g_c];
g_c +=1;

std::string combine_points_f = argv[g_c];//"cell_intensity_norm.npy";
g_c+=1;

printf("combine_points_f %s\n", combine_points_f.c_str());

int cpu_num = atoi(argv[g_c]);
g_c +=1;
printf("angle %s\n", angle.c_str());
printf("vb_r,  %d\n", vb_r);
printf("ants file %s\n", ants_dir_name.c_str());


int blockdim_x =atoi(argv[g_c]); //cp
g_c +=1;
int blockdim_y =atoi(argv[g_c]);  //N
g_c +=1;
int blockdim_z =atoi(argv[g_c]);  //N
g_c +=1;


int vx_num = x_b_num*y_b_num*z_b_num;
printf("total voxel  %d\n", vx_num);

printf("blockdim_z  %d\n", blockdim_z);

//read cell coordinates file
int CT_num = 48;
std::vector<int> CT_li;
for (int i = 0; i < CT_num ; i += 4) {
    CT_li.push_back(i);
}

std::vector<int> sample_ids;
for (int i = 1; i < 7 ; i += 1) {
    sample_ids.push_back(i);
}

// std::string sample = "CT0_01";

std::vector<std::string> exps{"1st", "2nd"} ;

//start timer
double iStart = cpuSecond();
//GPU device setup
int dev = 0;
hipDeviceProp_t deviceProp;
CHECK(hipGetDeviceProperties(&deviceProp, dev));
printf("Using Device %d: %s\n", dev, deviceProp.name);
CHECK(hipSetDevice(dev));

for (int l = 0; l < exps.size() ; l += 1) {
   
    std::string exp = exps[l];
    printf(" %s\n", exp.c_str());

    #pragma omp parallel for num_threads(cpu_num)
    for (int m = 0; m < CT_li.size() ; m += 1) {
        
        for (int n = 0; n < sample_ids.size() ; n += 1) {
            std::string pd = getLeftPaddingString(std::to_string(sample_ids[n]), 2, '0')  ;

            std::string sample = "CT"+std::to_string(CT_li[m])+ "_" + pd;
            printf(" %s\n", sample.c_str());

            std::string root_vx_f = savedir + "/" +exp + "/"+root_fol+"/"+std::to_string(vx)+"um/"+region+"/vb"+ std::to_string(vb_r)+"_mo"+std::to_string(mox)+"_"+std::to_string(moy)+"_"+std::to_string(moz)+"_r"+r_str+"/";
            std::filesystem::create_directories(root_vx_f); 

            std::string vx_f= root_vx_f + sample + "_vb_CT_"+exp+".bin";


            if (fileExists(vx_f)) {
                std::cout << vx_f << " exists" << std::endl;
            }else{


            std::vector<int> s;
            std::vector<double> cell_cords;  
            std::string cell_npy = savedir +"/"+ exp+"/" +  sample +"/SYTOX-G/"+ants_dir_name +"/"+combine_points_f;
            // std::cout << cell_npy << std::endl;
            
            aoba::LoadArrayFromNumpy(cell_npy, s, cell_cords);
            // std::cout << "size:" << s[0] << " " << s[1] << std::endl;

            // printf("i: %d ,  cell_cords[0]:%f\n", 0, cell_cords[0]);

            int cell_count = s[1];
            int col_num = s[0];

            // printf("col_num,  %d\n", col_num);
            // printf("cell_count,  %d\n", cell_count);

            // int *x=(int*)malloc(sizeof(int)*cell_count);
            // int *y=(int*)malloc(sizeof(int)*cell_count);
            // int *z=(int*)malloc(sizeof(int)*cell_count);

            std::vector<int> x(sizeof(int)*cell_count);
            std::vector<int> y(sizeof(int)*cell_count);
            std::vector<int> z(sizeof(int)*cell_count);
            // std::vector<int> intense(sizeof(int)*cell_count);


            for (int i = 0; i < cell_count ; i += 1) {
                for (int j = 0; j < col_num ; j += 1) {
                if (j==0){
                x[i] = static_cast<int>(cell_cords[j*cell_count+i]);
                }else if(j==1){
                y[i] = static_cast<int>(cell_cords[j*cell_count+i]);
                }else if(j==2){
                z[i] = static_cast<int>(cell_cords[j*cell_count+i]);
                } 
            }
            }

            // for (int i = 0; i < 1 ; i += 1) {
            //     printf("i: %d ,  x[i]:%d\n", i, x[i]);
            //     printf("i: %d ,  y[i]:%d\n", i, y[i]);
            //     printf("i: %d ,  z[i]:%d\n", i, z[i]);
                // printf("i: %d ,  intense[i]:%d\n", i, intense[i]);
            // }



            int *vx_count=(int*)malloc(sizeof(int)*vx_num);
            // double *vx_count_ratio=(double*)malloc(sizeof(double)*vx_num);

            // printf("a\n");
            // 


            // printf("aa\n");
            //device memory
            int *d_x,  *d_y, *d_z, *d_vx_count;
            CHECK(hipMalloc((void **)&d_x, sizeof(int)*cell_count));
            CHECK(hipMalloc((void **)&d_y, sizeof(int)*cell_count));
            CHECK(hipMalloc((void **)&d_z, sizeof(int)*cell_count));
            // CHECK(hipMalloc((void **)&d_intense, sizeof(int)*cell_count));
            CHECK(hipMalloc((void **)&d_vx_count, sizeof(int)*vx_num));


            // printf("aaa\n");

            //memcopy host -> device
            CHECK(hipMemcpy(d_x, x.data(), sizeof(int)*cell_count, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_y, y.data(), sizeof(int)*cell_count, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_z, z.data(), sizeof(int)*cell_count, hipMemcpyHostToDevice));    
            // CHECK(hipMemcpy(d_intense, intense.data(), sizeof(int)*cell_count, hipMemcpyHostToDevice));    
            
            CHECK(hipMemcpy(d_vx_count, vx_count, sizeof(int)*vx_num, hipMemcpyHostToDevice));   

                // int dimx =x_b_num/blockdim_x;// (NE+NI)/blockdim_x;
                // int dimy = y_b_num/blockdim_y;
                // int dimz = z_b_num/blockdim_z;
                // dim3 block(blockdim_x, blockdim_y, blockdim_z);
                // dim3 grid(dimx, dimy, dimz);
                blockdim_x=1; //x_b_num;
                blockdim_y=1;//y_b_num;
                blockdim_z=1;//z_b_num;

                // printf("blockdim_x: %d\n" ,blockdim_x);
                // printf("blockdim_y: %d\n" ,blockdim_y);
                // printf("blockdim_z: %d\n" ,blockdim_z);

                
                if (angle == "hor"){
                    int dimx =x_b_num/blockdim_x;// (NE+NI)/blockdim_x;
                    int dimy = y_b_num/blockdim_y;
                    int dimz = z_b_num/1;
                    dim3 block(blockdim_x, blockdim_y, 1);
                    dim3 grid(dimx, dimy, dimz);

                vx_gpu_hor<<<grid, block>>>(d_x, d_y,  d_z,  d_vx_count,  cell_count, x_b_num, y_b_num, z_b_num, xmin, ymin, zmin, xmax, ymax, zmax,  vx, vb_r, mox, moy, moz);
                }else if (angle == "cor"){
                    int dimx =x_b_num/blockdim_x;// (NE+NI)/blockdim_x;
                    int dimy = y_b_num/1;
                    int dimz = z_b_num/blockdim_z;
                    dim3 block(blockdim_x,1, blockdim_z);
                    dim3 grid(dimx, dimy, dimz);
                    vx_gpu_cor<<<grid, block>>>(d_x, d_y,  d_z, d_vx_count,  cell_count, x_b_num, y_b_num, z_b_num, xmin, ymin, zmin, xmax, ymax, zmax,  vx, vb_r, mox, moy, moz);
                }else if (angle == "sag"){
                    int dimx =x_b_num/1;// (NE+NI)/blockdim_x;
                    int dimy = y_b_num/blockdim_y;
                    int dimz = z_b_num/blockdim_z;
                    dim3 block(1, blockdim_y, blockdim_z);
                    dim3 grid(dimx, dimy, dimz);
                    vx_gpu_sag<<<grid, block>>>(d_x, d_y,  d_z, d_vx_count,  cell_count, x_b_num, y_b_num, z_b_num, xmin, ymin, zmin, xmax, ymax, zmax,  vx, vb_r, mox, moy, moz);
                }
                CHECK(hipDeviceSynchronize());
                CHECK(hipGetLastError());
                printf("end vx_count\n");
            
            

                        // copy kernel result back to host side
                
                CHECK(hipMemcpy(vx_count, d_vx_count, sizeof(int)*vx_num,  hipMemcpyDeviceToHost));


                // for (int i=0; i<vx_num; i++){
                    // vx_count_ratio[i] = static_cast<int>(vx_count[i]/cell_count);
                //     if (i<5){
                //     printf("i: %d ,  vx_count[i]:%d\n", i, vx_count[i]);
                //     }
                // }
                
            //savefile
            
            std::ofstream ofs;
                ofs.open(vx_f, std::ios::out|std::ios::binary|std::ios::trunc);
                if (!ofs) {
                std::cout << "Can't open a file"<<vx_f<<std::endl;
                }
                    
                for (int i=0; i<vx_num; i++){
                    ofs.write(( char * ) &vx_count[i],sizeof(int) );
                    // printf("count %d, cv %f\n", i, cvs[i]);
                    }//for
                ofs.close();

            CHECK(hipFree(d_x));
            CHECK(hipFree(d_y));
            CHECK(hipFree(d_z));
            // CHECK(hipFree(d_intense));
            CHECK(hipFree(d_vx_count));

            // free(x);
            // free(y);
            // free(z);
            free(vx_count);
            // free(vx_count_ratio);
// free(spc);
            }
        }
    }
   }

   //end timer
    double iElaps = cpuSecond() - iStart;
    printf("elapsed %f sec\n", iElaps);
}






// 

